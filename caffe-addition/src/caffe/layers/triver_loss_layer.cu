#include "hip/hip_runtime.h"
/*
* triver_loss_layer.cu
*
*/

#include <algorithm>
#include <vector>


#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/triver_loss_layer.hpp"


namespace caffe {

	template <typename Dtype>
	void TriVerLossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

		int count = bottom[0]->count() / 3;

		const Dtype * anchor_data = bottom[0]->gpu_data();
		const Dtype * pos_data = bottom[0]->gpu_data() + count;
		const Dtype * neg_data = bottom[0]->gpu_data() + 2 * count;

		const Dtype *label = bottom[1]->cpu_data();
		for (int i = 0; i < bottom[1]->num() / 3; i++)
		{
			CHECK_EQ(label[i], label[i + bottom[1]->num() / 3]);
			CHECK_NE(label[i], label[i + 2 * bottom[1]->num() / 3]);
		}

		//const Dtype* sampleW = bottom[3]->cpu_data();
		//const Dtype sampleW = Dtype(1);
		caffe_gpu_sub(
			count,
			anchor_data,  // a
			pos_data,  // p
			diff_ap_.mutable_gpu_data());  // a_i-p_i
		caffe_gpu_sub(
			count,
			anchor_data,  // a
			neg_data,  // n
			diff_an_.mutable_gpu_data());  // a_i-n_i
		caffe_gpu_sub(
			count,
			pos_data,  // p
			neg_data,  // n
			diff_pn_.mutable_gpu_data());  // p_i-n_i
		const int channels = bottom[0]->channels();

		caffe_gpu_powx(
			count,
			diff_ap_.mutable_gpu_data(),  // a_i-p_i
			Dtype(2),
			diff_sq_ap_.mutable_gpu_data());  // (a_i-p_i)^2
		caffe_gpu_gemv(
			CblasNoTrans,
			bottom[0]->num()/3,
			bottom[0]->channels(),
			Dtype(1.0),                                         //alpha
			diff_sq_ap_.gpu_data(),  // (a_i-p_i)^2                // A
			summer_vec_.gpu_data(),                             // x
			Dtype(0.0),                                         //belta
			dist_sq_ap_.mutable_gpu_data());  // \Sum (a_i-p_i)^2  //y

		caffe_gpu_powx(
			count,
			diff_an_.mutable_gpu_data(),  // a_i-n_i
			Dtype(2),
			diff_sq_an_.mutable_gpu_data());  // (a_i-n_i)^2
		caffe_gpu_gemv(
			CblasNoTrans,
			bottom[0]->num()/3,
			bottom[0]->channels(),
			Dtype(1.0),                                         //alpha
			diff_sq_an_.gpu_data(),  // (a_i-n_i)^2                // A
			summer_vec_.gpu_data(),                             // x
			Dtype(0.0),                                         //belta
			dist_sq_an_.mutable_gpu_data());  // \Sum (a_i-n_i)^2  //y


		//------------------------------------------------------------------------------
		//calculate m_0
		Dtype m_0 = 0;
		int  m_cnt = 0;
		for (int i = 0; i < bottom[0]->num() / 3; ++i)
		{
			Dtype thd = dist_sq_ap_.cpu_data()[i];
			int cnt = 0;
			for (int j = 0; j < bottom[0]->num() / 3; ++j)
			{
				if (dist_sq_ap_.cpu_data()[i] <= thd)
				{
					++cnt;
				}
				if (dist_sq_an_.cpu_data()[i] > thd)
				{
					++cnt;
				}
			}
			if (cnt > m_cnt)
			{
				m_cnt = cnt;
				m_0 = thd;
			}
		}

		for (int i = 0; i < bottom[0]->num() / 3; ++i)
		{
			Dtype thd = dist_sq_an_.cpu_data()[i];
			int cnt = 0;
			for (int j = 0; j < bottom[0]->num() / 3; ++j)
			{
				if (dist_sq_ap_.cpu_data()[i] <= thd)
				{
					++cnt;
				}
				if (dist_sq_an_.cpu_data()[i] > thd)
				{
					++cnt;
				}
			}
			if (cnt > m_cnt)
			{
				m_cnt = cnt;
				m_0 = thd;
			}
		}
		//-----------------------------------------------------------

		Dtype k = this->layer_param_.triver_loss_param().k();
		
		m_ = k*m_0;
		//m_ = 20;

		Dtype loss(0.0);
		for (int i = 0; i < bottom[0]->num() / 3; ++i) {
			Dtype mdist = std::max(m_ - dist_sq_an_.cpu_data()[i], Dtype(0.0));
			loss_an_.mutable_cpu_data()[i] = mdist;
			loss += dist_sq_ap_.cpu_data()[i] + mdist;
		}

		loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
		top[0]->mutable_cpu_data()[0] = loss;

	}

	template <typename Dtype>
	void TriVerLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		int num = bottom[0]->num() / 3;
		int channels = bottom[0]->channels();


		Dtype* anchor_out = bottom[0]->mutable_gpu_diff();
		Dtype* pos_out = bottom[0]->mutable_gpu_diff() + bottom[0]->count() / 3;
		Dtype* neg_out = bottom[0]->mutable_gpu_diff() + 2 * bottom[0]->count() / 3;

		Dtype lamda = this->layer_param_.triver_loss_param().lamda();

		const Dtype alpha = lamda * top[0]->cpu_diff()[0] /
			static_cast<Dtype>(bottom[0]->num());


		for (int j = 0; j < num; ++j)
		{
			if (loss_an_.cpu_data()[j] > Dtype(1e-9))
			{
				//anchor
				caffe_gpu_axpby(channels,
					alpha*Dtype(-1),
					diff_pn_.gpu_data() + (j*channels),
					Dtype(0.0),
					anchor_out + (j*channels));

				//positive
				caffe_gpu_axpby(channels,
					alpha*Dtype(-1),
					diff_ap_.gpu_data() + (j*channels),
					Dtype(0.0),
					pos_out + (j*channels));

				//negitive
				caffe_gpu_axpby(channels,
					alpha,
					diff_an_.gpu_data() + (j*channels),
					Dtype(0.0),
					neg_out + (j*channels));
			}
			else{
				//anchor
				caffe_gpu_axpby(channels,
					alpha,
					diff_ap_.gpu_data() + (j*channels),
					Dtype(0.0),
					anchor_out + (j*channels));

				//positive
				caffe_gpu_axpby(channels,
					alpha*Dtype(-1),
					diff_ap_.gpu_data() + (j*channels),
					Dtype(0.0),
					pos_out + (j*channels));

				//negitive
				caffe_gpu_set(channels,
					Dtype(0.0),
					neg_out + (j*channels));
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(TriVerLossLayer);

}  // namespace caffe
