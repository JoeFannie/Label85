#include "hip/hip_runtime.h"
/*
* triplet_loss_layer.cu
*
*/

#include <algorithm>
#include <vector>


#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/triplet_loss_layer.hpp"

namespace caffe {

  template <typename Dtype>
  void TripletLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	  int count = bottom[0]->count() / 3;

	  const Dtype * anchor_data = bottom[0]->gpu_data();
	  const Dtype * pos_data = bottom[0]->gpu_data() + count;
	  const Dtype * neg_data = bottom[0]->gpu_data() + 2 * count;

	  const Dtype *label = bottom[1]->cpu_data();
	  for (int i = 0; i < bottom[1]->num() / 3; i++)
	  {
		  CHECK_EQ(label[i], label[i + bottom[1]->num() / 3]);
		  CHECK_NE(label[i], label[i + 2 * bottom[1]->num() / 3]);
	  }

	  //const Dtype* sampleW = bottom[3]->cpu_data();
	  //const Dtype sampleW = Dtype(1);
	  caffe_gpu_sub(
		  count,
		  anchor_data,  // a
		  pos_data,  // p
		  diff_ap_.mutable_gpu_data());  // a_i-p_i
	  caffe_gpu_sub(
		  count,
		  anchor_data,  // a
		  neg_data,  // n
		  diff_an_.mutable_gpu_data());  // a_i-n_i
	  caffe_gpu_sub(
		  count,
		  pos_data,  // p
		  neg_data,  // n
		  diff_pn_.mutable_gpu_data());  // p_i-n_i
	  const int channels = bottom[0]->channels();

	  caffe_gpu_powx(
		  count,
		  diff_ap_.mutable_gpu_data(),  // a_i-p_i
		  Dtype(2),
		  diff_sq_ap_.mutable_gpu_data());  // (a_i-p_i)^2
	  caffe_gpu_gemv(
		  CblasNoTrans,
		  bottom[0]->num() / 3,
		  bottom[0]->channels(),
		  Dtype(1.0),                                         //alpha
		  diff_sq_ap_.gpu_data(),  // (a_i-p_i)^2                // A
		  summer_vec_.gpu_data(),                             // x
		  Dtype(0.0),                                         //belta
		  dist_sq_ap_.mutable_gpu_data());  // \Sum (a_i-p_i)^2  //y

	  caffe_gpu_powx(
		  count,
		  diff_an_.mutable_gpu_data(),  // a_i-n_i
		  Dtype(2),
		  diff_sq_an_.mutable_gpu_data());  // (a_i-n_i)^2
	  caffe_gpu_gemv(
		  CblasNoTrans,
		  bottom[0]->num() / 3,
		  bottom[0]->channels(),
		  Dtype(1.0),                                         //alpha
		  diff_sq_an_.gpu_data(),  // (a_i-n_i)^2                // A
		  summer_vec_.gpu_data(),                             // x
		  Dtype(0.0),                                         //belta
		  dist_sq_an_.mutable_gpu_data());  // \Sum (a_i-n_i)^2  //y

    Dtype margin = this->layer_param_.triplet_loss_param().margin();
    Dtype loss(0.0);

 

    for (int i = 0; i < bottom[0]->num()/3; ++i) {
		Dtype mdist = std::max(margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i], Dtype(0.0));
		loss += mdist;

		if (mdist < Dtype(1e-6)) {
			//prepare for backward pass
			caffe_gpu_set(channels, Dtype(0), diff_ap_.mutable_gpu_data() + (i*channels));
			caffe_gpu_set(channels, Dtype(0), diff_an_.mutable_gpu_data() + (i*channels));
			caffe_gpu_set(channels, Dtype(0), diff_pn_.mutable_gpu_data() + (i*channels));
		}
    }
    loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
    top[0]->mutable_cpu_data()[0] = loss;
  }

  template <typename Dtype>
  void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                             const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	  int num = bottom[0]->num() / 3;
	  int channels = bottom[0]->channels();


	  Dtype* anchor_out = bottom[0]->mutable_gpu_diff();
	  Dtype* pos_out = bottom[0]->mutable_gpu_diff() + bottom[0]->count() / 3;
	  Dtype* neg_out = bottom[0]->mutable_gpu_diff() + 2 * bottom[0]->count() / 3;

	  Dtype lamda = this->layer_param_.triplet_loss_param().lamda();

	  const Dtype alpha = lamda * top[0]->cpu_diff()[0] /
		  static_cast<Dtype>(bottom[0]->num());


	  for (int j = 0; j < num; ++j)
	  {
		  //anchor
		  caffe_gpu_axpby(channels,
				  alpha*Dtype(-1),
				  diff_pn_.gpu_data() + (j*channels),
				  Dtype(0.0),
				  anchor_out + (j*channels));

		  //positive
		  caffe_gpu_axpby(channels,
				  alpha*Dtype(-1),
				  diff_ap_.gpu_data() + (j*channels),
				  Dtype(0.0),
				  pos_out + (j*channels));

		  //negitive
		  caffe_gpu_axpby(channels,
				  alpha,
				  diff_an_.gpu_data() + (j*channels),
				  Dtype(0.0),
				  neg_out + (j*channels));
	  }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
